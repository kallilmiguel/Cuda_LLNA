#include<stdio.h>
#include<stdlib.h>
#include<stdbool.h>
#include<math.h>
#include<dirent.h>
#include"lib/graph.h"


#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include""
#include"lib/cuda_common.cuh"
#include"lib/tep.cuh"

int main(int argc, char* argv[]){

    char* graph_name = argv[1];
    const char* data_path = argv[2];
    const char* rule_path = argv[3];
    const char* output_path = argv[4];
    const char* seed_path = argv[5];

    int MAX_NODES = 12000;
    int steps=350;

    FILE* fseed = fopen(seed_path, "r");
    bool* init_state = (bool*) malloc(sizeof(bool)*MAX_NODES);
    for(int i=0;i<MAX_NODES;i++){
        fscanf(fseed, "%d\n", &init_state[i]);
    }
    
    printf("Iniciando grafo: %s\n", graph_name);

    char *file_path = (char*) malloc(sizeof(char)*100);
    strcpy(file_path,data_path);
    strcat(file_path, graph_name);
                
    Graph* G = read_adjList(file_path);
    
    printf("Nodes: %d, edges: %d\n", G->numVertices, G->numTransitions);
    //generate_dteps_selected(G, steps, dir->d_name, init_state, rule_path, output_path);
    generate_dteps_selected_statistics(G, steps, graph_name, init_state, rule_path, output_path);

    free(file_path);
    //freeGraph(G);
    free(G);

    //free(init_state);


    return 0;
}
