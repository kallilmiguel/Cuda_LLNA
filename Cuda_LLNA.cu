#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<stdbool.h>
#include<math.h>
#include<dirent.h>
#include"lib/graph.h"
//#include"lib/rules.h"
#include"lib/filesave.h"

#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include""
#include"lib/cuda_common.cuh"
#include"lib/statistics.cuh"
#include"lib/tep.cuh"


void generate_teps(Graph *G, int steps, char* file_name, bool* init_state);

__global__ void evolve_tep_gpu(bool* TEP, rules* bRules, int counterB, rules* sRules, float* density, 
double* resolution, int* degree, int* adjList, int* indexes, int* sum_of_states,
int rules_size, int number_of_nodes, int steps);

bool* createRandomInitState(int number_of_nodes);

arr_adjList* generate_gpu_adjlist(Graph* G);

int main(void){

    int steps=350;

    struct dirent *dir;
    const char *dir_path = "data/rulesel/";
    DIR *d;
    d = opendir(dir_path);
    int counter =1;
    int MAX_NODES = 2000;

    bool* init_state = createRandomInitState(MAX_NODES);

    FILE *chkp = fopen("data/checkpoint.txt", "r");
    int check;
    fscanf(chkp, "%d", &check);
    fclose(chkp);
    if(d){
        while((dir = readdir(d)) != NULL){
            if(!strcmp(get_filename_ext(dir->d_name), "txt")){
                if(counter >= check){
                    printf("Iniciando grafo %d: %s\n", counter, dir->d_name);
                
                    char *file_path = (char*) malloc(sizeof(char)*50);
                    strcpy(file_path,dir_path);
                    strcat(file_path, dir->d_name);

                    Graph* G = read_adjList(file_path);

                    generate_teps(G, steps, dir->d_name, init_state);

            
                    free(file_path);
                    free(G);
                }
                
                counter++;
                chkp = fopen("data/checkpoint.txt", "w");
                fprintf(chkp, "%d", counter); 
                fclose(chkp);

            }
            
        }
    }
    free(init_state);
    closedir(d);

    return 0;
}


bool* createRandomInitState(int number_of_nodes){
    bool* init_state = (bool*) malloc(sizeof(bool)*number_of_nodes);

    for(int i=0;i<number_of_nodes;i++){
        init_state[i] = rand()&1;
    }

    return init_state;
}

void generate_teps(Graph *G, int steps, char* file_name, bool* init_state){

    //in this function, we will use a number of threads equivalent to the number of survive rules
    //for example, for 8 neighbors, there will be 512 survive rules. Since a graph has n nodes and each node 
    //is counted as a thread, there will be 512*n threads. In this implementation, we will set as 512 the 
    //number of threads in each block and the number of blocks in each grid as n.

    //get size of rules
    int rules_size = 1;
    for(int i=0;i<NB_SIZE+1;i++){
        rules_size*=2;
    }

    //set block size as rules size and grid size as number of vertices
    dim3 block(rules_size);
    dim3 grid(G->numVertices);

    //get number of nodes
    int number_of_nodes = G->numVertices;

    //create birth and survive rule structure
    rules* bRules = getAllRules();
    rules* sRules = getAllRules();

    //allocate memory for gpu
    rules* gpu_bRules;
    rules* gpu_sRules;
    gpuErrchk(hipMalloc((void**)&gpu_bRules, sizeof(rules)*rules_size));
    gpuErrchk(hipMalloc((void**)&gpu_sRules, sizeof(rules)*rules_size));

    gpuErrchk(hipMemcpy(gpu_bRules, bRules, sizeof(rules)*rules_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_sRules, sRules, sizeof(rules)*rules_size, hipMemcpyHostToDevice));

    //free memory for host
    free(bRules);
    free(sRules);


    //resolution of densities to calculate the rules
    double* resolution = (double*)malloc(sizeof(double) * (NB_SIZE + 2));

    for (int i = 0; i < NB_SIZE + 2; i++) {
        resolution[i] = (i) / (double)(NB_SIZE + 1);
    }

    //transfer resolution value to device
    double* gpu_resolution;
    gpuErrchk(hipMalloc((void**)&gpu_resolution, sizeof(double)*(NB_SIZE+2)));
    gpuErrchk(hipMemcpy(gpu_resolution, resolution, sizeof(double)*(NB_SIZE+2), hipMemcpyHostToDevice));

    free(resolution);

    //calculate initial density for all nodes
    int* alive_neighbors = (int*) malloc(sizeof(int)*number_of_nodes);
    int* degree = (int*)malloc(sizeof(int)*number_of_nodes);
    float* density = (float*)malloc(sizeof(float)*number_of_nodes*rules_size);
    
    for(int i=0;i<G->numVertices;i++){
        alive_neighbors[i]=0;
        node *p = G->adjLists[i];
        degree[i]=0;
        while(p){
            if(init_state[p->vertex] == 1){
                alive_neighbors[i]+=1;
            }
            degree[i]++;
            p = p->next;
        }
    }

    for(int i=0;i<G->numVertices;i++){
        if(alive_neighbors[i]==0){
            for(int j=0;j<rules_size; j++){
                density[i+j*G->numVertices]=0;
            }
            
        }
        else{
            for(int j=0;j<rules_size; j++){
                if(degree[i] != 0){
                    density[i+j*G->numVertices] = (float)alive_neighbors[i]/(float)degree[i];
                }
                else{
                    density[i+j*G->numVertices] = 0;
                }
            }
        }
    }

    free(alive_neighbors);

    //since we are working with (number_of_nodes*number_of_survive_rules) number of threads, in order to reduce
    //the overhead we will reduce the number of copies of the TEP. We will only make one copy of the TEP at
    //every birth rule iteration. In order to do this, first we must allocate memory from the host, so that
    //we can copy the information from device to host.
    bool* TEP = (bool*) malloc(sizeof(bool)*number_of_nodes*steps*rules_size);
    
    //create array in device with same size as the host
    bool* gpu_TEP;
    
    gpuErrchk(hipMalloc((void**)&gpu_TEP, sizeof(bool)*number_of_nodes*steps*rules_size));
    
    //copy the initial state array to the TEP structure at the device, respecting the indexes
    for(int i=0;i<rules_size;i++){
        gpuErrchk(hipMemcpy(&gpu_TEP[i*G->numVertices*steps], &init_state[0], 
        sizeof(bool)*number_of_nodes, hipMemcpyHostToDevice));
    }

    //also, create arrays for both degree and density. Since density is dinamically updated, we will extend
    //the array to be the block size. Note that there is no need to do this for the degree array, since
    //it is a static array
    int* gpu_degree;
    float* gpu_density;
    
    gpuErrchk(hipMalloc((void**)&gpu_degree, sizeof(int)*number_of_nodes));
    gpuErrchk(hipMalloc((void**)&gpu_density, sizeof(float)*number_of_nodes*rules_size));

    //copy host values to device arrays
    gpuErrchk(hipMemcpy(gpu_degree, degree, sizeof(int)*number_of_nodes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_density, density, sizeof(float)*number_of_nodes*rules_size, hipMemcpyHostToDevice)); 
    //free memory for host arrays
    free(degree);

    //create two array structure on host representing the adjacencies from the graph
    int* adjList = (int*) malloc(sizeof(int)*2*G->numTransitions);
    int* indexes = (int*) malloc(sizeof(int)*G->numVertices);

    int counter=0;
    for(int i=0;i<G->numVertices;i++){
        indexes[i] = counter;
        node*p = G->adjLists[i];
        while(p){
            adjList[counter] = p->vertex;
            counter++;
            p = p->next;
        }
    }

    //and copy it to the device
    int* gpu_adjList;
    int* gpu_indexes;
    gpuErrchk(hipMalloc((void**)&gpu_adjList, sizeof(int)*2*G->numTransitions));
    gpuErrchk(hipMalloc((void**)&gpu_indexes, sizeof(int)*G->numVertices));
    gpuErrchk(hipMemcpy(gpu_adjList, adjList, sizeof(int)*2*G->numTransitions, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_indexes, indexes, sizeof(int)*G->numVertices, hipMemcpyHostToDevice));

    //free memory from host
    free(adjList);
    free(indexes);

    //create array to sum each cell state, in order to calculate shannon entropy
    int* gpu_sum_of_states;
    gpuErrchk(hipMalloc((void**)&gpu_sum_of_states, sizeof(int)*G->numVertices*rules_size));

    //define number of attributes in shannon entropy histogram
    int attributes = 20;
    
    //declare variable for shannon entropy histogram for boths host and device
    int* cpu_shannon_histogram = (int*)malloc(sizeof(int)*rules_size*rules_size*attributes);

    int* gpu_shannon_histogram;
    gpuErrchk(hipMalloc((void**)&gpu_shannon_histogram, sizeof(int)*rules_size*rules_size*attributes));
    gpuErrchk(hipMemset(gpu_shannon_histogram, 0, sizeof(int)*rules_size*rules_size*attributes));

    //declare variable for word entropy for both host and device
    int* cpu_word_histogram = (int*) malloc(sizeof(int)*rules_size*rules_size*attributes);

    int* gpu_word_histogram;
    gpuErrchk(hipMalloc((void**)&gpu_word_histogram, sizeof(int)*rules_size*rules_size*attributes));
    gpuErrchk(hipMemset(gpu_word_histogram, 0, sizeof(int)*rules_size*rules_size*attributes));

    //declare variable to calculate population on both host and device
    float* cpu_population = (float*) malloc(sizeof(float)*attributes*rules_size*rules_size);
    float* gpu_population;
    gpuErrchk(hipMalloc((void**)&gpu_population, sizeof(float)*attributes*rules_size*rules_size));
    gpuErrchk(hipMemset(gpu_population, 0, sizeof(float)*attributes*rules_size*rules_size));

    //declare variable to calculate two point correlation on both host and device
    float* cpu_correlation = (float*)malloc(sizeof(float)*attributes*rules_size*rules_size);
    float* gpu_correlation;
    gpuErrchk(hipMalloc((void**)&gpu_correlation, sizeof(float)*attributes*rules_size*rules_size));
    gpuErrchk(hipMemset(gpu_correlation, 0, sizeof(float)*attributes*rules_size*rules_size));

    //loop for B rules from 0 to 2^(Number of neighbors)
    for(int counterB=0;counterB<512;counterB++){
        //printf("\nBirth Rule Number: %d\n", counterB);
        
        evolve_tep_gpu <<<block, grid>>> (gpu_TEP, gpu_bRules, counterB, gpu_sRules, gpu_density, gpu_resolution,
        gpu_degree, gpu_adjList, gpu_indexes, gpu_sum_of_states, rules_size, G->numVertices, steps);
        gpuErrchk(hipMemcpy(gpu_density, density, sizeof(float)*number_of_nodes*rules_size, hipMemcpyHostToDevice));

        shannon_entropy <<<block, grid>>> (gpu_sum_of_states, steps, G->numVertices, counterB,
        gpu_shannon_histogram, attributes);
        gpuErrchk(hipDeviceSynchronize());

        population <<<block, grid>>> (gpu_TEP, steps, G->numVertices, counterB, rules_size, attributes, gpu_population);
        gpuErrchk(hipDeviceSynchronize());

        word_entropy_histogram <<<block, grid>>> (gpu_TEP, steps, G->numVertices, counterB, rules_size, attributes, 40, gpu_word_histogram);
        gpuErrchk(hipDeviceSynchronize());

        tp_correlation <<<block, steps>>> (gpu_TEP, steps, G->numVertices, counterB, rules_size, attributes, gpu_correlation);
        gpuErrchk(hipDeviceSynchronize());

    }

    //transfer shannon entropy histogram from device to host
    gpuErrchk(hipMemcpy(cpu_shannon_histogram, gpu_shannon_histogram, 
    sizeof(int)*rules_size*rules_size*attributes, hipMemcpyDeviceToHost));

    //transfer word entropy histogram from device to host
    gpuErrchk(hipMemcpy(cpu_word_histogram, gpu_word_histogram, 
    sizeof(int)*rules_size*rules_size*attributes, hipMemcpyDeviceToHost));

    //transfer population array from device to host
    gpuErrchk(hipMemcpy(cpu_population, gpu_population, sizeof(float)*rules_size*rules_size*attributes, hipMemcpyDeviceToHost));

    //transfer correlation array from device to host
    gpuErrchk(hipMemcpy(cpu_correlation, gpu_correlation, sizeof(float)*rules_size*rules_size*attributes, hipMemcpyDeviceToHost));

    //write file with results of shannon entropy
    save_csv_int(file_name,"data/measures/shannon/",cpu_shannon_histogram, attributes, rules_size*rules_size);

    //write file with results of word entropy
    save_csv_int(file_name,"data/measures/word/",cpu_word_histogram, attributes, rules_size*rules_size);

    //write file with results of population
    save_csv_float(file_name,"data/measures/population/",cpu_population, attributes, rules_size*rules_size);

    //write file with results of population
    save_csv_float(file_name,"data/measures/correlation/",cpu_correlation, attributes, rules_size*rules_size);

    //free remaining allocated memory from host and device
    free(TEP);
    free(cpu_shannon_histogram);
    hipFree(gpu_sum_of_states);
    hipFree(gpu_shannon_histogram);
    hipFree(gpu_TEP);
    hipFree(gpu_degree);
    hipFree(gpu_density);
    free(density);
    hipFree(gpu_resolution);
    hipFree(gpu_bRules);
    hipFree(gpu_sRules);
    hipFree(gpu_adjList);
    hipFree(gpu_indexes);
    free(cpu_population);
    free(cpu_correlation);
    hipFree(gpu_correlation);
    hipFree(gpu_population);
    free(cpu_word_histogram);
    hipFree(gpu_word_histogram);
}
